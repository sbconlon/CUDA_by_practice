#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include "Error.h"

__global__ void cubeKernel(float* d_out, float* d_in) {
    // -:YOUR CODE HERE:-
    size_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    d_out[tid] = d_in[tid] * d_in[tid] * d_in[tid];
}

void onDevice(float* h_in, float* h_out, int ARRAY_SIZE, int ARRAY_BYTES) {
    // declare GPU memory pointers
    // -:YOUR CODE HERE:-
    float *d_in, *d_out;

    // allocate GPU memory
    // -:YOUR CODE HERE:-
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_in, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_out, ARRAY_BYTES));

    // transfer the array to the GPU
    // -:YOUR CODE HERE:-
    HANDLER_ERROR_ERR(hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice));

    // launch the kernel
    cubeKernel<<<1, ARRAY_SIZE>>>(d_out, d_in);

    // copy back the result array to the CPU
    // -:YOUR CODE HERE:-
    HANDLER_ERROR_ERR(hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost));

    // free GPU memory pointers
    // -:YOUR CODE HERE:-
    HANDLER_ERROR_ERR(hipFree(d_in));
    HANDLER_ERROR_ERR(hipFree(d_out));
}

void test(float* h_in, float* h_out, int ARRAY_SIZE, int ARRAY_BYTES) {
    // print out the resulting array
    for (int i = 0; i < ARRAY_SIZE; i++) {
        printf("%d^3 = %f\n", i, h_out[i]);
        assert(h_out[i] == (h_in[i] * h_in[i] * h_in[i]));
        //printf("%f", h_out[i]);
        //printf(((i % 4) != 3) ? "\t" : "\n");
    }

    printf("-: successful execution :-\n");
}

void onHost() {
    const int ARRAY_SIZE = 64;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    // declare CPU memory pointers
    // -:YOUR CODE HERE:-
    float *h_in, *h_out;


    // Allocate CPU memory pointers
    // -:YOUR CODE HERE:-
    h_in = (float*)malloc(ARRAY_BYTES);
    h_out = (float*)malloc(ARRAY_BYTES);

    for (int i = 0; i < ARRAY_SIZE; i++) {
        h_in[i] = float(i);
    }

    // call the kernel
    onDevice(h_in, h_out, ARRAY_SIZE, ARRAY_BYTES);
    test(h_in, h_out, ARRAY_SIZE, ARRAY_BYTES);

    // free CPU memory pointers
    // -:YOUR CODE HERE:-
    free(h_in);
    free(h_out);
}

int main(int argc, char** argv) {
    onHost();

    return 0;
}